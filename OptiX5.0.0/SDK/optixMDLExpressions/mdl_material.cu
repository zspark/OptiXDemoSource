#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "mdl_structs.h"
#include "phong.h"

using namespace optix;


rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_tangent,  attribute shading_tangent, );
rtDeclareVariable(float3, texcoord,         attribute texcoord, );

rtDeclareVariable(float, reflection_coefficient, , );


rtDeclareVariable(
    rtCallableProgramId<
        void(float3 *, MDL_SDK_State const *, MDL_SDK_Res_data_pair const *, void *)>,
    mdl_expr,,);

rtDeclareVariable(
    rtCallableProgramId<
        void(float3 *, MDL_SDK_State const *, MDL_SDK_Res_data_pair const *, void *)>,
    mdl_shading_normal_expr,,);

rtDeclareVariable(
    rtCallableProgramId<
        void(float3 *, MDL_Environment_state const *, MDL_SDK_Res_data_pair const *, void *)>,
    mdl_env_expr,,);


// computes direct lighting using phong shading
RT_PROGRAM void closest_hit_radiance()
{
    float3 world_shading_normal =
        normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ));
    float3 world_geometric_normal =
        normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ));

    //
    // Initialize state for MDL
    //

    float world_to_object[16];
    float object_to_world[16];
    rtGetTransform(RT_WORLD_TO_OBJECT, world_to_object);
    rtGetTransform(RT_OBJECT_TO_WORLD, object_to_world);

    float3 hit_point = ray.origin + t_hit * ray.direction;

    float3 text_coords = texcoord;      // taking an address of an attribute is not supported
    float3 tangent_u = shading_tangent;
    float3 tangent_v = cross(world_shading_normal, tangent_u);

    MDL_SDK_State state = {
        world_shading_normal,
        world_geometric_normal,
        hit_point,
        0,                           // animation_time
        &text_coords,                // *text_coords;
        &tangent_u,                  // *tangent_u;
        &tangent_v,                  // *tangent_v;
        NULL,                        // *text_results;
        NULL,                        // *ro_data_segment;
        (float4 *) world_to_object,
        (float4 *) object_to_world,
        0                            // object_id
    };

    MDL_SDK_Res_data_pair res_data_pair = {
        NULL,
        NULL
    };

    //
    // Calculate shading normal
    //

    mdl_shading_normal_expr(&world_shading_normal, &state, &res_data_pair, NULL);
    state.normal = world_shading_normal;

    //
    // Calculate tint
    //

    float3 tint;
    mdl_expr(&tint, &state, &res_data_pair, NULL);

    //
    // Shade with phong shading
    //

    float lum = luminanceCIE(tint);
    float specular_factor = 1.f - 0.4 * lum;
    phongShade(
        tint,                                 // diffuse
        make_float3(0.01f),                   // ambient
        make_float3(specular_factor),         // specular
        make_float3(reflection_coefficient),  // reflection
        80.f,                                 // phong exponent
        world_shading_normal);
}

RT_PROGRAM void any_hit_shadow()
{
    phongShadowed();
}


RT_PROGRAM void miss()
{
    //
    // Initialize state for MDL
    //

    MDL_Environment_state state = {
        optix::normalize(ray.direction)
    };

    MDL_SDK_Res_data_pair res_data_pair = {
        NULL,
        NULL
    };

    //
    // Calculate environment color
    //

    mdl_env_expr(&prd.result, &state, &res_data_pair, NULL);
}


